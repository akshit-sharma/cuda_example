#include "hip/hip_runtime.h"
#include "header_with_function.hpp"
#include "header_with_kernel.hpp"

#include <iostream>

void some_function() {
  std::cout << "some_function in file_with_kernel.cu" << std::endl;
}

__global__ void set_value_via_kernel(int *dest, int value) { dest[0] = value; }

void SomeClass::set_value(int value) {
  set_value_via_kernel<<<1, 1>>>(data, value);
  hipDeviceSynchronize();
}

int SomeClass::get_value() {
  int return_value = 0;
  hipMemcpy(&return_value, data, sizeof(int),
             hipMemcpyKind::hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  return return_value;
}
